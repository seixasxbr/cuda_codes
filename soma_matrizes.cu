
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 9

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) 
{
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); 
    a[0]=1;
    a[1]=2;
    a[2]=3;
    a[3]=4;
    a[4]=5;
    a[5]=6;
    a[6]=7;
    a[7]=8;
    a[8]=9;
    // for (int i=0;i<N;i++) {
	// 	printf("a[%d]=%d\n",i,a[i]);
	// }
	b = (int *)malloc(size);
    b[0]=2;
    b[1]=3;
    b[2]=4;
    b[3]=5;
    b[4]=6;
    b[5]=7;
    b[6]=8;
    b[7]=9;
    b[8]=10;
	c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// Launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup

	for (int i=0;i<N;i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
	}
    // 
    int M[3][3];
    M[0][0] = c[0];
    M[0][1] = c[1];
    M[0][2] = c[2];
    M[1][0] = c[3];
    M[1][1] = c[4];
    M[1][2] = c[5];
    M[2][0] = c[6];
    M[2][1] = c[7];
    M[2][2] = c[8];
    printf("\n");
    printf("MATRIZ DE RESULTADO:\n");
    printf("M[0][0]=%d, M[0][1]=%d, M[0][2]=%d\nM[1][0]=%d, M[1][1]=%d, M[1][2]=%d\nM[2][0]=%d, M[2][1]=%d, M[2][2]=%d\n",M[0][0],M[0][1],M[0][2],M[1][0],M[1][1],M[1][2],M[2][0],M[2][1],M[2][2]);
    printf("\n");
    for (int j=0;j<3;j++){
        for (int i=0;i<3;i++) {
            int index=j*3+i;
            M[j][i] = c[index];
            printf("M[%d][%d]=%d ",j,i,M[j][i]);
        }
        printf("\n");
    }
    // for (int i=0;i<3;i++) {
    //     M[0][i] = c[i];
	// 	printf("%d ",M[0][i]);
	// }
    
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}